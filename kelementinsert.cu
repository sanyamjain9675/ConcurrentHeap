#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include <time.h>

//total size of the heap
#define maxSize 1000

__global__ void Insert_Elem(volatile int *heap,int *d_elements,int *curSize,volatile int *lockArr,int *elemSize,int k){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
    {
        int childInd = atomicInc((unsigned *) curSize,maxSize+10);
	    childInd = childInd*k;
        heap[childInd] = d_elements[tid*k];
	    heap[childInd] = d_elements[tid*k+1];
	

        int parInd = ((childInd/k - 1)/2) * k;
        int oldval = 1;
        do
        {
            oldval = atomicCAS((int*)&lockArr[parInd/k],0,1);
            if(oldval == 0) //if we got the lock on parent
            {
                if(heap[parInd] > heap[childInd])
                {
		            for(int i = 0;i<k;i++){
                    	int temp = heap[parInd+i];    //swapping the elements
                    	heap[parInd+i] = heap[childInd+i];
                    	heap[childInd+i] = temp;
		            }

                    __threadfence();//necessary

                    lockArr[childInd/k] = 0; //unlock the child
    
                    childInd = parInd;
                    parInd = ((childInd/k - 1)/2) * k;
                    oldval = 1; //we need to heapify again

                    //if we have reached the root
                    if(childInd == 0){
                        oldval = 0; //we need not heapify again
                        lockArr[childInd/k] = 0;
                    }  
                }
                else //if heap property satisfied release the locks
                {
                    lockArr[childInd/k] = 0;
                    lockArr[parInd/k] = 0;
                } 
                
            }
            // __threadfence(); //doesnt seem necessary
        }while(oldval != 0);
    }
}
bool checkHeap(int *ar,int size,int k)
{
    for(int i = 0;i<size/2;i+=k)
    {
        if(ar[i] > ar[2*i + k]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + k);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + k]); 
            return false;
        } 
        if((2*i + 2) < size && ar[i] > ar[2*i + 2*k]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 2*k);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 2*k]);
            return false;
        }
    }
    return true;
}
int getRandom(int lower, int upper)
{
    int num = (rand() % (upper - lower + 1)) + lower;
    return num;  
}
void printArray(int arr[],int size,int k)
{
    for(int i = 0;i<size;i++)
        printf("%d, ",arr[i]);
}
void FillArray(int elements[],int size,int k)
{
    for(int i = 0;i<size*k;i++)
    {
        elements[i] = getRandom(1,1000);
    }
}
    
void heapify(int hp[],int ind,int size,int k)
{
    while(1)
    {
        int leftChild = 2*ind+k;
        int rightChild = 2*ind+2*k;
        int largeInd = -1;
        if(rightChild < size*k && hp[ind] > hp[rightChild]){
            if(hp[leftChild] < hp[rightChild])
                largeInd = leftChild;
            else
                largeInd = rightChild;
        }
        else if(leftChild < size*k && hp[ind] > hp[leftChild]){
            largeInd = leftChild;
        }
        
        if(largeInd == -1)  return;
	
        

	for(int i = 0;i<k;i++){
		int temp = hp[ind+i];
        	hp[ind+i] = hp[largeInd+i];
        	hp[largeInd+i] = temp;
	}

	ind = largeInd;
        
    }

}

void buildHeap(int hp[],int n,int k)
{
    for(int i = n/2 -1 ; i>=0;i--)
    {
        heapify(hp,i*k,n,k);
    }
}

__global__ void setLockVar(int *curSize,int *lockArr,int *elemSize)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
        lockArr[tid + *curSize] = 1;
}

double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main() {
    srand(time(0));
    int countvalid = 0;
    int inivalid = 0;
    int k = 3;
    
    for(int lk = 0;lk<100;lk++)
    {
        int *d_a;
        int *curSize;
        int *lockArr;
        int *elemSize;
        hipHostAlloc(&curSize, sizeof(int), 0);
        hipHostAlloc(&elemSize, sizeof(int), 0);

        int h_a[maxSize*k];
        *curSize = getRandom(1,maxSize/10);

        //Initialise Heap with some random values
        FillArray(h_a,*curSize,k);

       //heapify the heap
        buildHeap(h_a,*curSize,k);

       //check if satisfies the heap property
        if(checkHeap(h_a,*curSize,k)) inivalid++;

        hipMalloc(&d_a,maxSize*k*sizeof(int)); 
        hipMemcpy(d_a,h_a,maxSize*k*sizeof(int),hipMemcpyHostToDevice);

        *elemSize = getRandom(1,maxSize-*curSize-2);
        int elements[*elemSize*k];
        
        FillArray(elements,*elemSize,k);
        printf("%d. No of Inserted Elements are = %d\n",inivalid,*elemSize);

        int *d_elements;
        hipMalloc(&d_elements,*elemSize*k*sizeof(int));
        hipMemcpy(d_elements,elements,*elemSize * k* sizeof(int),hipMemcpyHostToDevice);
        hipMalloc(&lockArr,(*elemSize + *curSize)*sizeof(int));
        hipMemset(lockArr,0,(*elemSize + *curSize)*sizeof(int));
    
        int block = ceil((float) *elemSize/1024);

        double starttime = rtclock(); 
        setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
        hipDeviceSynchronize();
        Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
        hipDeviceSynchronize();
        double endtime = rtclock();  
        printtime("GPU Kernel time: ", starttime, endtime);
        hipMemcpy(h_a,d_a,maxSize*k*sizeof(int),hipMemcpyDeviceToHost);
        
        if(checkHeap(h_a,*curSize,k)) {
            // printf("Valid\n");
            countvalid++;
        }
    }
    printf("\nInitial valid : %d",inivalid);
    printf("\nvalid : %d\n\n",countvalid);
    return 0;
}
