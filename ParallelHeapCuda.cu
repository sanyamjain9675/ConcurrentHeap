    #include <cstdio>        // Added for printf() function 
    #include <sys/time.h>    // Added to get time of day
    #include <hip/hip_runtime.h>
    #include <fstream>
    #include <time.h>
    #include <iostream>

    //total size of the heap
    #define maxSize 1000000

    __global__ void Insert_Elem(int *heap,int *d_elements,int *curSize,int *elemSize,int k){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
        {
            heap[tid + *curSize] = d_elements[tid];
        }
    }

    __global__ void delete_Elem(int *heap,int *d_elements,int *curSize,int *elemSize,int k){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
        {
            heap[tid + *curSize] = d_elements[tid];
        }
    }

    bool checkHeap(int *ar,int size,int k)
    {
        for(int i = 0;i<size/2;i+=k)
        {
            if(ar[i] > ar[2*i + k]){
                printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + k);
                printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + k]); 
                return false;
            } 
            if((2*i + 2) < size && ar[i] > ar[2*i + 2*k]){
                printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 2*k);
                printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 2*k]);
                return false;
            }
        }
        return true;
    }

    int getRandom(int lower, int upper)
    {
        int num = (rand() % (upper - lower + 1)) + lower;
        return num;  
    }

    void printArray(int arr[],int size,int k)
    {
        printf("\n");
        for(int i = 0;i<size;i++)
            printf("%d, ",arr[i]);
        
        printf("\n");
    }

    void FillArray(int elements[],int size,int k)
    {
        for(int i = 0;i<size*k;i++)
        {
            elements[i] = getRandom(1,1000);
        }
    }
        
    void heapify(int hp[],int ind,int size,int k)
    {
        while(1)
        {
            int leftChild = 2*ind+k;
            int rightChild = 2*ind+2*k;
            int largeInd = -1;
            if(rightChild < size*k && hp[ind] > hp[rightChild]){
                if(hp[leftChild] < hp[rightChild])
                    largeInd = leftChild;
                else
                    largeInd = rightChild;
            }
            else if(leftChild < size*k && hp[ind] > hp[leftChild]){
                largeInd = leftChild;
            }
            
            if(largeInd == -1)  return;
        
            

        for(int i = 0;i<k;i++){
            int temp = hp[ind+i];
                hp[ind+i] = hp[largeInd+i];
                hp[largeInd+i] = temp;
        }

        ind = largeInd;
            
        }

    }

    void buildHeap(int hp[],int n,int k)
    {
        for(int i = n/2 -1 ; i>=0;i--)
        {
            heapify(hp,i*k,n,k);
        }
    }

    __global__ void setLockVar(int *curSize,int *lockArr,int *elemSize)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
            lockArr[tid + *curSize] = 1;
    }

    __device__ void heapifyBUP(int arr[], int n, int childInd, int k) {
        // Find parent 
        int parInd = ((childInd/k - 1)/2) * k;
        if (parInd >= 0) { 
            if (arr[childInd] < arr[parInd]) { 
                for(int i = 0;i<k;i++){
                    int temp = arr[parInd+i];
                    arr[parInd+i] = arr[childInd+i];
                    arr[childInd+i] = temp;
                }
                heapifyBUP(arr, n, parInd,k); 
            } 
        } 
    }

    __device__ void insertNode(int arr[],  int *n,int val,int k)
    {
        // Increase the size of Heap by 2
        *n = *n + 1;
        int childInd = *n * k;
    
        // Insert the element at end of Heap
        // arr[childInd - 2] = Key;
        arr[childInd - 1] = val;
    
        // Heapify the new node following a
        // Bottom-up approach
        heapifyBUP(arr, *n,childInd-k,k);
    }

    //(serHeap,*serSize,*elements,*elemSize);
    __global__ void insertNodeHelper(int *arr,int *size,int *elements,int *elemSize)
    {
        int k = 1;
        for(int i = 0;i<*elemSize;i++){
            insertNode(arr,size,elements[i],k);
        }
    }


    double rtclock(){
        struct timezone Tzp;
        struct timeval Tp;
        int stat;
        stat = gettimeofday(&Tp, &Tzp);
        if (stat != 0) printf("Error return from gettimeofday: %d", stat);
        return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
    }

    void printtime(const char *str, double starttime, double endtime){
        printf("%s%3f seconds\n", str, endtime - starttime);
    }

    int main() {
        srand(time(0));
        int countvalid = 0,newValid = 0,inivalid = 0,k = 1;
        int *d_a,*curSize,*lockArr,*elemSize,*serSize,*serHeap;

        hipHostAlloc(&curSize, sizeof(int), 0);
        hipHostAlloc(&elemSize, sizeof(int), 0);
        hipHostAlloc(&serSize, sizeof(int), 0);

        int newHeap[maxSize*k];
        int h_a[maxSize*k];

        *curSize = getRandom(1,maxSize/10);
        *serSize = *curSize;

        //Initialise Heap with some random values
        FillArray(h_a,*curSize,k);

        //heapify the heap
        //buildHeap(h_a,*curSize,k);

        //check if satisfies the heap property
        //if(checkHeap(h_a,*curSize,k)) inivalid++;

        hipMalloc(&d_a,maxSize*sizeof(int)); 
        hipMalloc(&serHeap,maxSize*sizeof(int)); 

        hipMemcpy(d_a,h_a,maxSize * sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(serHeap,h_a,maxSize * sizeof(int),hipMemcpyHostToDevice);

        for(int lk = 0;lk<10;lk++)
        {
            do{
                *elemSize = getRandom(1,maxSize-*curSize-2);
            }while(*elemSize + *curSize > maxSize);
            
            int elements[*elemSize*k];
            
            FillArray(elements,*elemSize,k);

            printf("%d. No of Inserted Elements are = %d\n",inivalid,*elemSize);

            int *d_elements;
            hipMalloc(&d_elements,*elemSize*k*sizeof(int));
            hipMemcpy(d_elements,elements,*elemSize * k* sizeof(int),hipMemcpyHostToDevice);
            // cudaMalloc(&lockArr,(*elemSize + *curSize)*sizeof(int));
            // cudaMemset(lockArr,0,(*elemSize + *curSize)*sizeof(int));
        
            int block = ceil((float) *elemSize/1024);

            double starttime = rtclock(); 
            // setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
            // cudaDeviceSynchronize();
            Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,elemSize,k);
            hipDeviceSynchronize();
            double endtime = rtclock();  
            printtime("GPU Kernel time: ", starttime, endtime);

            // starttime = rtclock();
            // insertNodeHelper<<<1,1>>>(serHeap,serSize,d_elements,elemSize);
            // cudaDeviceSynchronize();
            // endtime = rtclock();
            // printtime("GPU (1 thread time)Kernel time: ", starttime, endtime);
            
            hipMemcpy(h_a,d_a,maxSize*k*sizeof(int),hipMemcpyDeviceToHost);
            //cudaMemcpy(newHeap,serHeap,maxSize*k*sizeof(int),cudaMemcpyDeviceToHost);
            // if(checkHeap(h_a,*curSize,k)) {
            //     // printf("Valid\n");
            //     countvalid++;
            // }

            // if(checkHeap(newHeap,*serSize,k)) {
            //     // printf("Valid\n");
            //     newValid++;
            // }
            
        }

        // printf("\nInitial valid : %d",inivalid);
        // printf("\nSingle Thread : %d",newValid);
        // printf("\nMulti Thread  : %d",countvalid);
        printf( " Over ");
        return 0;
    }
