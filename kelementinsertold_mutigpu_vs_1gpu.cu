    #include <cstdio>        // Added for printf() function 
    #include <sys/time.h>    // Added to get time of day
    #include <hip/hip_runtime.h>
    #include <fstream>
    #include <time.h>
    #include <iostream>

    //total size of the heap
    #define maxSize 100

    __global__ void Insert_Elem(volatile int *heap,int *d_elements,int *curSize,volatile int *lockArr,int *elemSize,int k){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
        {
            int childInd = atomicInc((unsigned *) curSize,maxSize);
            childInd = childInd*k;
            for(int i = 0;i<k;i++)
                heap[childInd+i] = d_elements[tid*k+i];

            int parInd = ((childInd/k - 1)/2) * k;

            if(childInd == 0){
                lockArr[childInd] = 0;
            }

            if(childInd != 0)
            {
                int oldval = 1;
                do
                {
                    oldval = atomicCAS((int*)&lockArr[parInd/k],0,1);
                    if(oldval == 0) //if we got the lock on parent
                    {
                        if(heap[parInd] > heap[childInd])
                        {
                            for(int i = 0;i<k;i++){
                                int temp = heap[parInd+i];    //swapping the elements
                                heap[parInd+i] = heap[childInd+i];
                                heap[childInd+i] = temp;
                            // __threadfence();//necessary
                            }
                            
                            __threadfence();

                            lockArr[childInd/k] = 0; //unlock the child
            
                            childInd = parInd;
                            parInd = ((childInd/k - 1)/2) * k;
                            oldval = 1; //we need to heapify again

                            //if we have reached the root
                            if(childInd == 0){
                                oldval = 0; //we need not heapify again
                                lockArr[childInd/k] = 0;
                            }  
                        }
                        else //if heap property satisfied release the locks
                        {
                            lockArr[childInd/k] = 0;
                            lockArr[parInd/k] = 0;
                        } 
                        
                    }
                    //__threadfence(); //doesnt seem necessary
                }while(oldval != 0);
            }
        }
    }

    bool checkHeap(int *ar,int size,int k)
    {
        for(int i = 0;i<size/2;i+=k)
        {
            if(ar[i] > ar[2*i + k]){
                printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + k);
                printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + k]); 
                return false;
            } 
            if((2*i + 2) < size && ar[i] > ar[2*i + 2*k]){
                printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 2*k);
                printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 2*k]);
                return false;
            }
        }
        return true;
    }
    int getRandom(int lower, int upper)
    {
        int num = (rand() % (upper - lower + 1)) + lower;
        return num;  
    }
    void printArray(int arr[],int size,int k)
    {
        printf("\n");
        for(int i = 0;i<size;i++)
            printf("%d, ",arr[i]);
        
        printf("\n");
    }
    void FillArray(int elements[],int size,int k)
    {
        for(int i = 0;i<size*k;i++)
        {
            elements[i] = getRandom(1,1000);
        }
    }
        
    void heapify(int hp[],int ind,int size,int k)
    {
        while(1)
        {
            int leftChild = 2*ind+k;
            int rightChild = 2*ind+2*k;
            int largeInd = -1;
            if(rightChild < size*k && hp[ind] > hp[rightChild]){
                if(hp[leftChild] < hp[rightChild])
                    largeInd = leftChild;
                else
                    largeInd = rightChild;
            }
            else if(leftChild < size*k && hp[ind] > hp[leftChild]){
                largeInd = leftChild;
            }
            
            if(largeInd == -1)  return;
        
            

        for(int i = 0;i<k;i++){
            int temp = hp[ind+i];
                hp[ind+i] = hp[largeInd+i];
                hp[largeInd+i] = temp;
        }

        ind = largeInd;
            
        }

    }

    void buildHeap(int hp[],int n,int k)
    {
        for(int i = n/2 -1 ; i>=0;i--)
        {
            heapify(hp,i*k,n,k);
        }
    }

    __global__ void setLockVar(int *curSize,int *lockArr,int *elemSize)
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < *elemSize)
            lockArr[tid + *curSize] = 1;
    }

    __device__ void heapifyBUP(int arr[], int n, int childInd, int k) {
        // Find parent 
        int parInd = ((childInd/k - 1)/2) * k;
        if (parInd >= 0) { 
            if (arr[childInd] < arr[parInd]) { 
                for(int i = 0;i<k;i++){
                    int temp = arr[parInd+i];
                    arr[parInd+i] = arr[childInd+i];
                    arr[childInd+i] = temp;
                }
                heapifyBUP(arr, n, parInd,k); 
            } 
        } 
    }

    __device__ void insertNode(int arr[],  int *n,int val,int k)
    {
        // Increase the size of Heap by 2
        *n = *n + 1;
        int childInd = *n * k;
    
        // Insert the element at end of Heap
        // arr[childInd - 2] = Key;
        arr[childInd - 1] = val;
    
        // Heapify the new node following a
        // Bottom-up approach
        heapifyBUP(arr, *n,childInd-k,k);
    }

    //(serHeap,*serSize,*elements,*elemSize);
    __global__ void insertNodeHelper(int *arr,int *size,int *elements,int *elemSize)
    {
        int k = 1;
        for(int i = 0;i<*elemSize;i++){
            insertNode(arr,size,elements[i],k);
        }
    }


    double rtclock(){
        struct timezone Tzp;
        struct timeval Tp;
        int stat;
        stat = gettimeofday(&Tp, &Tzp);
        if (stat != 0) printf("Error return from gettimeofday: %d", stat);
        return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
    }

    void printtime(const char *str, double starttime, double endtime){
        printf("%s%3f seconds\n", str, endtime - starttime);
    }

    int main() {
        srand(time(0));
        int countvalid = 0;
        int newValid = 0;
        int inivalid = 0;
        int k = 1;
        
        for(int lk = 0;lk<10;lk++)
        {
            int *d_a;
            int *curSize;
            int *lockArr;
            int *elemSize;
            int *serHeap;
            int  *serSize;
            int newHeap[maxSize*k];

            hipHostAlloc(&curSize, sizeof(int), 0);
            hipHostAlloc(&elemSize, sizeof(int), 0);
            hipHostAlloc(&serSize, sizeof(int), 0);

            int h_a[maxSize*k];
            *curSize = getRandom(1,maxSize/10);
            *serSize = *curSize;

            //Initialise Heap with some random values
            FillArray(h_a,*curSize,k);

        //heapify the heap
            buildHeap(h_a,*curSize,k);

        //check if satisfies the heap property
            if(checkHeap(h_a,*curSize,k)) inivalid++;

            hipMalloc(&d_a,maxSize*sizeof(int)); 
            hipMalloc(&serHeap,maxSize*sizeof(int)); 

            hipMemcpy(d_a,h_a,maxSize * sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy(serHeap,h_a,maxSize * sizeof(int),hipMemcpyHostToDevice);

            *elemSize = getRandom(1,maxSize-*curSize-2);
            int elements[*elemSize*k];
            
            FillArray(elements,*elemSize,k);
            printf("%d. No of Inserted Elements are = %d\n",inivalid,*elemSize);

            int *d_elements;
            hipMalloc(&d_elements,*elemSize*k*sizeof(int));
            hipMemcpy(d_elements,elements,*elemSize * k* sizeof(int),hipMemcpyHostToDevice);
            hipMalloc(&lockArr,(*elemSize + *curSize)*sizeof(int));
            hipMemset(lockArr,0,(*elemSize + *curSize)*sizeof(int));
        
            int block = ceil((float) *elemSize/1024);

            double starttime = rtclock(); 
            setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
            hipDeviceSynchronize();
            Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
            hipDeviceSynchronize();
            double endtime = rtclock();  
            printtime("GPU Kernel time: ", starttime, endtime);

            starttime = rtclock();
            insertNodeHelper<<<1,1>>>(serHeap,serSize,d_elements,elemSize);
            hipDeviceSynchronize();
            endtime = rtclock();
            printtime("GPU (1 thread time)Kernel time: ", starttime, endtime);
            
            hipMemcpy(h_a,d_a,maxSize*k*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(newHeap,serHeap,maxSize*k*sizeof(int),hipMemcpyDeviceToHost);
            if(checkHeap(h_a,*curSize,k)) {
                // printf("Valid\n");
                countvalid++;
            }

            if(checkHeap(newHeap,*serSize,k)) {
                // printf("Valid\n");
                newValid++;
            }
            // bool flag = 0;
            // for(int i = 0;i<*curSize;i++){
            //     if(h_a[i] != newHeap[i]){
            //         flag = 1;
            //         break;
            //     }
            // }

            // if(flag == 0)   newValid++;
            // else    printf("Not Equal\n");

            // printArray(h_a,*curSize,k);
            // printArray(newHeap,*serSize,k);
        }
        printf("\nInitial valid : %d",inivalid);
        printf("\nSingle Thread : %d",newValid);
        printf("\nMulti Thread  : %d",countvalid);
        return 0;
    }
