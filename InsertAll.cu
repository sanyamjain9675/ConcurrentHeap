#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include <time.h>

//total size of the heap
#define maxSize 1000

__global__ void Insert_Elem(volatile int *heap,int *d_elements,int *curSize,volatile int *lockArr,int *elemSize){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
    {
        int childInd = atomicInc((unsigned *) curSize,maxSize+10);
        heap[childInd] = d_elements[tid];

        int parInd = (childInd-1)/2;

        if(childInd == 0){
            lockArr[childInd] = 0;
        }

        if(childInd != 0)
        {
            int oldval = 1;
            do
            {
                oldval = atomicCAS((int*)&lockArr[parInd],0,1);
                if(oldval == 0) //if we got the lock on parent
                {
                    if(heap[parInd] > heap[childInd])
                    {
                        int temp = heap[parInd];    //swapping the elements
                        heap[parInd] = heap[childInd];
                        heap[childInd] = temp;

                        __threadfence();//necessary

                        lockArr[childInd] = 0; //unlock the child
        
                        childInd = parInd;
                        parInd = (childInd-1)/2;
                        oldval = 1; //we need to heapify again

                        //if we have reached the root
                        if(childInd == 0){
                            oldval = 0; //we need not heapify again
                            lockArr[childInd] = 0;
                        }  
                    }
                    else //if heap property satisfied release the locks
                    {
                        lockArr[childInd] = 0;
                        lockArr[parInd] = 0;
                    } 
                    
                }
                // __threadfence(); //doesnt seem necessary
            }while(oldval != 0);
        }
    }
}
bool checkHeap(int *ar,int size)
{
    for(int i = 0;i<size/2;i++)
    {
        if(ar[i] > ar[2*i + 1]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 1);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 1]); 
            return false;
        } 
        if((2*i + 2) < size && ar[i] > ar[2*i + 2]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 2);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 2]);
            return false;
        }
    }
    return true;
}
int getRandom(int lower, int upper)
{
    int num = (rand() % (upper - lower + 1)) + lower;
    return num;  
}
void printArray(int arr[],int size)
{
    for(int i = 0;i<size;i++)
        printf("%d, ",arr[i]);
}
void FillArray(int elements[],int size)
{
    for(int i = 0;i<size;i++)
    {
        elements[i] = getRandom(1,1000);
    }
}
    
void heapify(int hp[],int ind,int size)
{
    while(1)
    {
        int leftChild = 2*ind+1;
        int rightChild = 2*ind+2;
        int largeInd = -1;
        if(rightChild < size && hp[ind] > hp[rightChild]){
            if(hp[leftChild] < hp[rightChild])
                largeInd = leftChild;
            else
                largeInd = rightChild;
        }
        else if(leftChild < size && hp[ind] > hp[leftChild]){
            largeInd = leftChild;
        }
        
        if(largeInd == -1)  return;
        int temp = hp[ind];
        hp[ind] = hp[largeInd];
        hp[largeInd] = temp;
        ind = largeInd;
    }

}

void buildHeap(int hp[],int n)
{
    for(int i = n/2 -1 ; i>=0;i--)
    {
        heapify(hp,i,n);
    }
}

__global__ void setLockVar(int *curSize,int *lockArr,int *elemSize)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
        lockArr[tid + *curSize] = 1;
}

double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main() {
    srand(time(0));
    int countvalid = 0;
    int inivalid = 0;
    
    for(int lk = 0;lk<100;lk++)
    {
        int *d_a;
        int *curSize;
        int *lockArr;
        int *elemSize;
        hipHostAlloc(&curSize, sizeof(int), 0);
        hipHostAlloc(&elemSize, sizeof(int), 0);

        int h_a[maxSize];
        // *curSize = getRandom(1,maxSize/10);
        *curSize = 0;

        //Initialise Heap with some random values
        FillArray(h_a,*curSize);

       //heapify the heap
        buildHeap(h_a,*curSize);

       //check if satisfies the heap property
        if(checkHeap(h_a,*curSize)) inivalid++;

        hipMalloc(&d_a,maxSize*sizeof(int)); 
        hipMemcpy(d_a,h_a,maxSize * sizeof(int),hipMemcpyHostToDevice);

        *elemSize = getRandom(1,maxSize-*curSize-2);
        int elements[*elemSize];
        
        FillArray(elements,*elemSize);
        printf("%d. No of Inserted Elements are = %d\n",inivalid,*elemSize);

        int *d_elements;
        hipMalloc(&d_elements,*elemSize*sizeof(int));
        hipMemcpy(d_elements,elements,*elemSize * sizeof(int),hipMemcpyHostToDevice);
        hipMalloc(&lockArr,(*elemSize + *curSize)*sizeof(int));
        hipMemset(lockArr,0,(*elemSize + *curSize)*sizeof(int));
    
        int block = ceil((float) *elemSize/1024);

        double starttime = rtclock(); 
        setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
        hipDeviceSynchronize();
        Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize);
        hipDeviceSynchronize();
        double endtime = rtclock();  
        printtime("GPU Kernel time: ", starttime, endtime);
        hipMemcpy(h_a,d_a,maxSize*sizeof(int),hipMemcpyDeviceToHost);
        
        if(checkHeap(h_a,*curSize)) {
            // printf("Valid\n");
            countvalid++;
        }
    }
    printf("\nInitial valid : %d",inivalid);
    printf("\nvalid : %d\n\n",countvalid);
    return 0;
}
